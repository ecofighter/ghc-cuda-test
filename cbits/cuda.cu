#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define CHECK(call) \
{ \
  const hipError_t error = call; \
  if(error != hipSuccess) \
  { \
    printf("Error: %s: %d, ", __FILE__, __LINE__); \
    printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
    exit(1); \
  } \
}

extern "C" __global__ void dotp(double* A, double* B, double* out){
  int bx = blockIdx.x;
  int bdx = blockDim.x;
  int gdx = gridDim.x;
  int tx =  threadIdx.x;
  int BLOCK_SIZE = 32;

  double a[20], b[20];

  /*** Global A,B -> Register a,b ***/
#pragma unroll
  for(int i = 0; i < 20; i++){
    a[i] = A[20*BLOCK_SIZE*bx + tx + BLOCK_SIZE*i];
  }

#pragma unroll
  for(int i = 0; i < 20; i++){
    b[i] = B[20*BLOCK_SIZE*bx + tx + BLOCK_SIZE*i];
  }

  /**** dot production ***/
  double o = 0;
  for(int i = 0; i < 20; ++i){
    o += a[i]*b[i];
  }

  /**** Register o -> Global out ***/
  out[bx*BLOCK_SIZE + tx] = o;
}

extern "C" void dot(int size){
  int N = size;
  int T = 32;
//  int T = atoi(argv[2]);

  double * A, *B, *out;
  A = (double*)malloc( N*N*N*20*sizeof(double));
  B = (double*)malloc( N*N*N*20*sizeof(double) );
  out = (double*)malloc( N*N*N*sizeof(double));


  // initialize
  for(int i = 0; i < N*N*N; ++i){
    for(int j = 0; j < 20; ++j){
      A[i*20 + j] = 100*i+j;
      B[i*20 + j] = 1000*i+j;
    }
    out[i] = 0.0;
  }


  double *dA, *dB, *dout;
  hipMalloc( (void**)&dA, N*N*N*20*sizeof(double));
  hipMalloc( (void**)&dB, N*N*N*20*sizeof(double));
  hipMalloc( (void**)&dout, N*N*N*sizeof(double));

  hipMemcpy(A, dA, N*N*N*20*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(B, dB, N*N*N*20*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(out, dout, N*N*N*sizeof(double), hipMemcpyHostToDevice);

  dim3 grid(N*N*N/T);
  dim3 block(T);

//  StartTimer();
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
  dotp<<<grid,block>>>(dA,dB,dout);
  CHECK(hipDeviceSynchronize());

hipEventRecord(stop,0);
hipEventSynchronize(stop);
float time;
hipEventElapsedTime(&time, start, stop);

//  double time = GetTimer(); // [ms]
  double flops = 39*N*N*N / (time * 1e-3); // Flop/sec
  printf("%d^3: time %f[ms], flops %f [GFlops]\n", N, time, flops * 1e-9);

  hipMemcpy(A, dA, N*N*N*20*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(B, dB, N*N*N*20*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(out, dout, N*N*N*sizeof(double), hipMemcpyDeviceToHost);

  free(out);
  free(A);
  free(B);

  hipFree(dout);
  hipFree(dA);
  hipFree(dB);
}
